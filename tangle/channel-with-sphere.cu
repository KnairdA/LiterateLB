#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include "util/volumetric_example.h"
#include "sampler/velocity_norm.h"
#include "sampler/curl_norm.h"
#include "sampler/q_criterion.h"

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

int main() {
hipSetDevice(0);

const descriptor::Cuboid<DESCRIPTOR> cuboid(300, 80, 80);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
  if (p.z == 0 || p.z == cuboid.nZ-1) {
    return 2; // boundary cell
  } else if (p.y == 0 || p.y == cuboid.nY-1) {
    return 3; // boundary cell
  } else if (p.x == 0) {
    return 4; // inflow cell
  } else if (p.x == cuboid.nX-1) {
    return 5; // outflow cell
  } else {
    return 1; // bulk
  }
});

for (std::size_t iX=0; iX < cuboid.nX; ++iX) {
  materials.set(gid(cuboid, iX, 0,           0), 6);
  materials.set(gid(cuboid, iX, cuboid.nY-1, 0), 6);
  materials.set(gid(cuboid, iX, 0,           cuboid.nZ-1), 6);
  materials.set(gid(cuboid, iX, cuboid.nY-1, cuboid.nZ-1), 6);
}

auto obstacle = [cuboid] __host__ __device__ (float3 p) -> float {
                  float3 q = p - make_float3(cuboid.nX/6, cuboid.nY/2, cuboid.nZ/2);
                  return sdf::sphere(q, cuboid.nY/T{5});
                };
materials.sdf(obstacle, 0);
SignedDistanceBoundary bouzidi(lattice, materials, obstacle, 1, 0);

auto bulk_mask    = materials.mask_of_material(1);
auto wall_mask_z  = materials.mask_of_material(2);
auto wall_mask_y  = materials.mask_of_material(3);
auto inflow_mask  = materials.mask_of_material(4);
auto outflow_mask = materials.mask_of_material(5);
auto edge_mask    = materials.mask_of_material(6);

lattice.apply(Operator(InitializeO(), bulk_mask),
              Operator(InitializeO(), wall_mask_z),
              Operator(InitializeO(), wall_mask_y),
              Operator(InitializeO(), inflow_mask),
              Operator(InitializeO(), outflow_mask),
              Operator(InitializeO(), edge_mask));

hipDeviceSynchronize();

VolumetricExample renderer(cuboid);
renderer.add<QCriterionS>(lattice, bulk_mask, obstacle);
renderer.add<CurlNormS>(lattice, bulk_mask, obstacle);
renderer.add<VelocityNormS>(lattice, bulk_mask, obstacle);
renderer.run([&](std::size_t iStep) {
  const float tau = 0.51;
  const float inflow = 0.08;
  
  lattice.apply(Operator(BgkCollideO(), bulk_mask, tau),
                Operator(BounceBackFreeSlipO(), wall_mask_z, WallNormal<0,0,1>()),
                Operator(BounceBackFreeSlipO(), wall_mask_y, WallNormal<0,1,0>()),
                Operator(EquilibriumVelocityWallO(), inflow_mask, std::min(iStep*1e-4, 1.0)*inflow, WallNormal<1,0,0>()),
                Operator(EquilibriumDensityWallO(), outflow_mask, 1, WallNormal<-1,0,0>()),
                Operator(BounceBackO(), edge_mask));
  lattice.apply<BouzidiO>(bouzidi.getCount(), bouzidi.getConfig());
  
  lattice.stream();
});
}
