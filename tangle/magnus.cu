#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include <LLBM/kernel/collect_moments.h>
#include <LLBM/kernel/collect_velocity_norm.h>

using T = float;
using DESCRIPTOR = descriptor::D2Q9;

int main() {
hipSetDevice(0);

const descriptor::Cuboid<DESCRIPTOR> cuboid(1200, 500);
Lattice<DESCRIPTOR,T> lattice(cuboid);

const float tau = 0.54;
const float u_inflow = 0.02;
const float u_rotate = 0.08;

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint2 p) -> int {
  if (p.x == 0) {
    return 3; // inflow
  } else if (p.x == cuboid.nX-1) {
    return 4; // outflow
  } else if (p.y == 0 || p.y == cuboid.nY-1) {
    return 2; // wall
  } else {
    return 1; // bulk
  }
});

materials.set(gid(cuboid, 0,0), 2);
materials.set(gid(cuboid, 0,cuboid.nY-1), 2);
materials.set(gid(cuboid, cuboid.nX-1,0), 5);
materials.set(gid(cuboid, cuboid.nX-1,cuboid.nY-1), 5);

auto cylinder = [cuboid] __host__ __device__ (float2 p) -> float {
                  float2 q = p - make_float2(cuboid.nX/6, 3*cuboid.nY/4);
                  float2 r = p - make_float2(cuboid.nX/6, 1*cuboid.nY/4);
                  return sdf::add(sdf::sphere(q, cuboid.nY/18),
                                  sdf::sphere(r, cuboid.nY/18));
                };

materials.sdf(cylinder, 0);
SignedDistanceBoundary bouzidi(lattice, materials, cylinder, 1, 0);

bouzidi.setVelocity([cuboid,u_rotate](float2 p) -> float2 {
  float2 q = p - make_float2(cuboid.nX/6, 3*cuboid.nY/4);
  if (length(q) < 1.1*cuboid.nY/18) {
    return u_rotate * normalize(make_float2(-q.y, q.x));
  } else {
    return make_float2(0);
  }
});

auto bulk_mask = materials.mask_of_material(1);
auto wall_mask = materials.mask_of_material(2);
auto inflow_mask  = materials.mask_of_material(3);
auto outflow_mask = materials.mask_of_material(4);
auto edge_mask = materials.mask_of_material(5);

lattice.apply(Operator(InitializeO(), bulk_mask),
              Operator(InitializeO(), wall_mask),
              Operator(InitializeO(), inflow_mask),
              Operator(InitializeO(), outflow_mask),
              Operator(InitializeO(), edge_mask));
hipDeviceSynchronize();

RenderWindow window("Magnus");
hipSurfaceObject_t colormap;
ColorPalette palette(colormap);
auto slice = [cuboid] __device__ (int iX, int iY) -> std::size_t {
               return descriptor::gid(cuboid,iX,cuboid.nY-1-iY);
             };
DeviceBuffer<T> moments_rho(cuboid.volume);
DeviceBuffer<T> moments_u(2*cuboid.volume);
T* u = moments_u.device();
std::size_t iStep = 0;

while (window.isOpen()) {
  lattice.apply(Operator(BgkCollideO(), bulk_mask, tau),
                Operator(BounceBackFreeSlipO(), wall_mask, WallNormal<0,1>()),
                Operator(EquilibriumVelocityWallO(), inflow_mask, std::min(iStep*1e-5, 1.)*u_inflow, WallNormal<1,0>()),
                Operator(EquilibriumDensityWallO(), outflow_mask, 1., WallNormal<-1,0>()),
                Operator(BounceBackO(), edge_mask));
  lattice.apply<BouzidiO>(bouzidi.getCount(), bouzidi.getConfig());
  lattice.stream();
  if (iStep % 100 == 0) {
    hipDeviceSynchronize();
    lattice.inspect<CollectMomentsF>(bulk_mask, moments_rho.device(), moments_u.device());
    renderSliceViewToTexture<<<
      dim3(cuboid.nX / 32 + 1, cuboid.nY / 32 + 1),
      dim3(32,32)
    >>>(cuboid.nX, cuboid.nY,
        slice,
        [u,u_rotate] __device__ (std::size_t gid) -> float {
          return length(make_float2(u[2*gid+0], u[2*gid+1])) / u_rotate;
        },
        [colormap] __device__ (float x) -> float3 {
          return colorFromTexture(colormap, clamp(x, 0.f, 1.f));
        },
        window.getRenderSurface());
    window.draw([&]() {
      ImGui::Begin("Render");
      palette.interact();
      ImGui::End();
    }, [](sf::Event&) { });
  }
  ++iStep;
}
}
