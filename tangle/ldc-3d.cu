#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include "util/volumetric_example.h"
#include "sampler/velocity_norm.h"
#include "sampler/curl_norm.h"
#include "sampler/shear_layer.h"

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

int main() {
if (cuda::device::count() == 0) {
  std::cerr << "No CUDA devices on this system" << std::endl;
  return -1;
}
auto current = cuda::device::current::get();

const descriptor::Cuboid<DESCRIPTOR> cuboid(100, 100, 100);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
  if (p.x == 0 || p.x == cuboid.nX-1 || p.y == 0 || p.y == cuboid.nY-1 || p.z == cuboid.nZ-1) {
    return 2; // boundary cell
  } else if (p.z == 0) {
    return 3; // lid cell
  } else {
    return 1; // bulk
  }
});

auto bulk_mask = materials.mask_of_material(1);
auto wall_mask = materials.mask_of_material(2);
auto lid_mask  = materials.mask_of_material(3);

cuda::synchronize(current);

auto none = [] __device__ (float3) -> float { return 1; };
VolumetricExample renderer(cuboid);
renderer.add<CurlNormS>(lattice, bulk_mask, none);
renderer.add<ShearLayerVisibilityS>(lattice, bulk_mask, none, make_float3(0,1,0));
renderer.add<VelocityNormS>(lattice, bulk_mask, none);
renderer.run([&](std::size_t iStep) {
  const float tau = 0.56;
  const float lid = 0.10;
  
  lattice.apply(Operator(BgkCollideO(), bulk_mask, tau),
                Operator(BounceBackO(), wall_mask),
                Operator(BounceBackMovingWallO(), lid_mask, std::min(iStep*1e-3, 1.0)*lid, 0.f, 0.f));
  
  lattice.stream();
});
}
