#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include "util/volumetric_example.h"
#include "sampler/velocity_norm.h"
#include "sampler/curl_norm.h"
#include "sampler/q_criterion.h"

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

int main() {
hipSetDevice(0);

const descriptor::Cuboid<DESCRIPTOR> cuboid(500, 80, 80);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
  if (p.y == 0 || p.y == cuboid.nY-1 || p.z == 0 || p.z == cuboid.nZ-1) {
    return 2; // boundary cell
  } else if (p.x == 0) {
    return 3; // inflow cell
  } else if (p.x == cuboid.nX-1) {
    return 4; // outflow cell
  } else {
    return 1; // bulk
  }
});

auto obstacle = [cuboid] __host__ __device__ (float3 p) -> float {
                  float3 q = p - make_float3(cuboid.nX/24.2f, cuboid.nY/2, cuboid.nZ/2);
                  return sdf::ssub(sdf::sphere(make_float2(q.y,q.z), cuboid.nY/T{9}),
                                   sdf::box(q, make_float3(cuboid.nX/128,cuboid.nY/2,cuboid.nZ/2)),
                                   5);
                };
materials.sdf(obstacle, 0);
SignedDistanceBoundary bouzidi(lattice, materials, obstacle, 1, 0);

auto bulk_mask     = materials.mask_of_material(1);
auto boundary_mask = materials.mask_of_material(2);
auto inflow_mask   = materials.mask_of_material(3);
auto outflow_mask  = materials.mask_of_material(4);

lattice.apply(Operator(InitializeO(), bulk_mask),
              Operator(InitializeO(), boundary_mask),
              Operator(InitializeO(), inflow_mask),
              Operator(InitializeO(), outflow_mask));

hipDeviceSynchronize();

VolumetricExample renderer(cuboid);
renderer.add<CurlNormS>(lattice, bulk_mask, obstacle);
renderer.add<QCriterionS>(lattice, bulk_mask, obstacle);
renderer.add<VelocityNormS>(lattice, bulk_mask, obstacle);
renderer.run([&](std::size_t iStep) {
  const float tau = 0.501;
  const float smagorinsky = 0.1;
  const float inflow = 0.0075;
  
  lattice.apply(Operator(SmagorinskyBgkCollideO(), bulk_mask, tau, smagorinsky),
                Operator(BounceBackO(), boundary_mask),
                Operator(EquilibriumVelocityWallO(), inflow_mask, std::min(iStep*1e-4, 1.0)*inflow, WallNormal<1,0,0>()),
                Operator(EquilibriumDensityWallO(), outflow_mask, 1, WallNormal<-1,0,0>()));
  lattice.apply<BouzidiO>(bouzidi.getCount(), bouzidi.getConfig());
  
  lattice.stream();
});
}
