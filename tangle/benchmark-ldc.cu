#include "hip/hip_runtime.h"
#include <LLBM/base.h>

#include <LLBM/kernel/collide.h>
#include <LLBM/kernel/bounce_back.h>
#include <LLBM/kernel/bounce_back_moving_wall.h>

#include "util/timer.h"

#include <iostream>

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

void simulate(descriptor::Cuboid<DESCRIPTOR> cuboid, std::size_t nStep) {
  hipSetDevice(0);

  Lattice<DESCRIPTOR,T> lattice(cuboid);
  
  CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
    if (p.x == 0 || p.x == cuboid.nX-1 || p.y == 0 || p.y == cuboid.nY-1 || p.z == 0) {
      return 2; // boundary cell
    } else if (p.z == cuboid.nZ-1) {
      return 3; // lid cell
    } else {
      return 1; // bulk
    }
  });
  
  auto bulk_mask = materials.mask_of_material(1);
  auto box_mask  = materials.mask_of_material(2);
  auto lid_mask  = materials.mask_of_material(3);
  
  hipDeviceSynchronize();

  for (std::size_t iStep=0; iStep < 100; ++iStep) {
    lattice.apply(Operator(BgkCollideO(), bulk_mask, 0.56),
                  Operator(BounceBackO(), box_mask),
                  Operator(BounceBackMovingWallO(), lid_mask, 0.05f, 0.f, 0.f));
    lattice.stream();
  }

  hipDeviceSynchronize();

  auto start = timer::now();

  for (std::size_t iStep=0; iStep < nStep; ++iStep) {
    lattice.apply(Operator(BgkCollideO(), bulk_mask, 0.56),
                  Operator(BounceBackO(), box_mask),
                  Operator(BounceBackMovingWallO(), lid_mask, 0.05f, 0.f, 0.f));
    lattice.stream();
  }

  hipDeviceSynchronize();

  auto mlups = timer::mlups(cuboid.volume, nStep, start);

  std::cout << sizeof(T) << ", " << cuboid.nX << ", " << nStep << ", " << mlups << std::endl;
}

int main(int argc, char* argv[]) {
  if (argc != 3) {
    std::cerr << "Invalid parameter count" << std::endl;
    return -1;
  }

  const std::size_t n     = atoi(argv[1]);
  const std::size_t steps = atoi(argv[2]);

  simulate({ n, n, n}, steps);

  return 0;
}
