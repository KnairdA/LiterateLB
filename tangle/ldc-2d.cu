#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include <LLBM/kernel/collect_moments.h>
#include <LLBM/kernel/collect_velocity_norm.h>

using T = float;
using DESCRIPTOR = descriptor::D2Q9;

int main() {
hipSetDevice(0);

const descriptor::Cuboid<DESCRIPTOR> cuboid(500, 500);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint2 p) -> int {
  if (p.x == 0 || p.y == 0 || p.x == cuboid.nX-1) {
    return 2; // boundary cell
  } else if (p.y == cuboid.nY-1) {
    return 3; // lid cell
  } else {
    return 1; // bulk
  }
});

auto bulk_mask = materials.mask_of_material(1);
auto wall_mask = materials.mask_of_material(2);
auto lid_mask  = materials.mask_of_material(3);

lattice.apply(Operator(InitializeO(), bulk_mask),
              Operator(InitializeO(), wall_mask),
              Operator(InitializeO(), lid_mask));
hipDeviceSynchronize();

const float tau = 0.51;
const float u_lid = 0.05;

RenderWindow window("LDC");
hipSurfaceObject_t colormap;
ColorPalette palette(colormap);
auto slice = [cuboid] __device__ (int iX, int iY) -> std::size_t {
               return descriptor::gid(cuboid,iX,cuboid.nY-1-iY);
             };
DeviceBuffer<T> moments_rho(cuboid.volume);
DeviceBuffer<T> moments_u(2*cuboid.volume);
T* u = moments_u.device();
std::size_t iStep = 0;

while (window.isOpen()) {
  lattice.apply(Operator(BgkCollideO(), bulk_mask, tau),
                Operator(BounceBackO(), wall_mask),
                Operator(BounceBackMovingWallO(), lid_mask, std::min(iStep*1e-3, 1.0)*u_lid, 0.f));
  lattice.stream();
  if (iStep % 100 == 0) {
    hipDeviceSynchronize();
    lattice.inspect<CollectMomentsF>(bulk_mask, moments_rho.device(), moments_u.device());
    renderSliceViewToTexture<<<
      dim3(cuboid.nX / 32 + 1, cuboid.nY / 32 + 1),
      dim3(32,32)
    >>>(cuboid.nX, cuboid.nY,
        slice,
        [u,u_lid] __device__ (std::size_t gid) -> float {
          return length(make_float2(u[2*gid+0], u[2*gid+1])) / u_lid;
        },
        [colormap] __device__ (float x) -> float3 {
          return colorFromTexture(colormap, clamp(x, 0.f, 1.f));
        },
        window.getRenderSurface());
    window.draw([&]() {
      ImGui::Begin("Render");
      palette.interact();
      ImGui::End();
    }, [](sf::Event&) { });
  }
  ++iStep;
}
}
