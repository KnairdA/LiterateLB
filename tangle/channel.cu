#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include "util/volumetric_example.h"
#include "sampler/velocity_norm.h"
#include "sampler/curl_norm.h"
#include "sampler/q_criterion.h"

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

int main() {
if (cuda::device::count() == 0) {
  std::cerr << "No CUDA devices on this system" << std::endl;
  return -1;
}
auto current = cuda::device::current::get();

const descriptor::Cuboid<DESCRIPTOR> cuboid(500, 100, 100);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
  if (p.z == 0 || p.z == cuboid.nZ-1) {
    return 2; // boundary cell
  } else if (p.y == 0 || p.y == cuboid.nY-1) {
    return 3; // boundary cell
  } else if (p.x == 0) {
    return 4; // inflow cell
  } else if (p.x == cuboid.nX-1) {
    return 5; // outflow cell
  } else {
    return 1; // bulk
  }
});

for (std::size_t iX=0; iX < cuboid.nX; ++iX) {
  materials.set(gid(cuboid, iX, 0,           0), 6);
  materials.set(gid(cuboid, iX, cuboid.nY-1, 0), 6);
  materials.set(gid(cuboid, iX, 0,           cuboid.nZ-1), 6);
  materials.set(gid(cuboid, iX, cuboid.nY-1, cuboid.nZ-1), 6);
}

auto obstacle = [cuboid] __host__ __device__ (float3 p) -> float {
  p -= make_float3(cuboid.nX/5, cuboid.nY/2, cuboid.nZ/2);
  float3 q = sdf::twisted(p, 0.01);
  return sdf::sphere(p, cuboid.nY/3.5) + sin(0.2*q.x)*sin(0.2*q.y)*sin(0.2*q.z);
};

materials.sdf(obstacle, 0);
SignedDistanceBoundary bouzidi(lattice, materials, obstacle, 1, 0);

auto bulk_mask    = materials.mask_of_material(1);
auto wall_mask_z  = materials.mask_of_material(2);
auto wall_mask_y  = materials.mask_of_material(3);
auto inflow_mask  = materials.mask_of_material(4);
auto outflow_mask = materials.mask_of_material(5);
auto edge_mask    = materials.mask_of_material(6);

cuda::synchronize(current);

VolumetricExample renderer(cuboid);
renderer.add<QCriterionS>(lattice, bulk_mask, obstacle);
renderer.add<CurlNormS>(lattice, bulk_mask, obstacle);
renderer.add<VelocityNormS>(lattice, bulk_mask, obstacle);
renderer.run([&](std::size_t iStep) {
  const float tau = 0.501;
  const float smagorinsky = 0.1;
  const float inflow = 0.04;
  
  lattice.apply(Operator(SmagorinskyBgkCollideO(), bulk_mask, tau, smagorinsky),
                Operator(BounceBackFreeSlipO(), wall_mask_z, WallNormal<0,0,1>()),
                Operator(BounceBackFreeSlipO(), wall_mask_y, WallNormal<0,1,0>()),
                Operator(EquilibriumVelocityWallO(), inflow_mask, std::min(iStep*1e-4, 1.0)*inflow, WallNormal<1,0,0>()),
                Operator(EquilibriumDensityWallO(), outflow_mask, 1, WallNormal<-1,0,0>()),
                Operator(BounceBackO(), edge_mask));
  lattice.apply<BouzidiO>(bouzidi.getCount(), bouzidi.getConfig());
  
  lattice.stream();
});
}
