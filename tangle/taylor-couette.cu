#include "hip/hip_runtime.h"
#include <LLBM/base.h>
#include <LLBM/bulk.h>
#include <LLBM/boundary.h>

#include "util/render_window.h"
#include "util/texture.h"
#include "util/colormap.h"

#include "util/volumetric_example.h"
#include "sampler/velocity_norm.h"
#include "sampler/curl_norm.h"
#include "sampler/shear_layer.h"

using T = float;
using DESCRIPTOR = descriptor::D3Q19;

int main() {
hipSetDevice(0);

const descriptor::Cuboid<DESCRIPTOR> cuboid(500, 96, 96);
Lattice<DESCRIPTOR,T> lattice(cuboid);

CellMaterials<DESCRIPTOR> materials(cuboid, [&cuboid](uint3 p) -> int {
  if (p.x == 0 || p.x == cuboid.nX-1) {
    return 2;
  } else {
    return 1;
  }
});

auto inner_cylinder = [cuboid] __host__ __device__ (float3 p) -> float {
                        float3 q = p - make_float3(0, cuboid.nY/2, cuboid.nZ/2);
                        return sdf::sphere(make_float2(q.y,q.z), cuboid.nY/T{4.5});
                      };
auto geometry = [cuboid,inner_cylinder] __host__ __device__ (float3 p) -> float {
                  float3 q = p - make_float3(0, cuboid.nY/2, cuboid.nZ/2);
                  return sdf::add(-sdf::sphere(make_float2(q.y,q.z), cuboid.nY/T{2.14}), inner_cylinder(p));
                };
materials.sdf(geometry, 0);
SignedDistanceBoundary bouzidi(lattice, materials, geometry, 1, 0);

const float wall = 0.2;

bouzidi.setVelocity([cuboid,wall](float3 p) -> float3 {
  float3 q = p - make_float3(0, cuboid.nY/2, cuboid.nZ/2);
  if (length(make_float2(q.y,q.z)) < cuboid.nY/T{2.5}) {
    return wall * normalize(make_float3(0, -q.z, q.y));
  } else {
    return make_float3(0);
  }
});

auto bulk_mask = materials.mask_of_material(1);
auto bulk_list = materials.list_of_material(1);
auto wall_mask = materials.mask_of_material(2);
auto wall_list = materials.list_of_material(2);

lattice.apply<InitializeO>(bulk_list);
lattice.apply<InitializeO>(wall_list);

hipDeviceSynchronize();

VolumetricExample renderer(cuboid);
renderer.add<VelocityNormS>(lattice, bulk_mask, inner_cylinder);
renderer.add<ShearLayerVisibilityS>(lattice, bulk_mask, inner_cylinder, make_float3(1,0,0));
renderer.run([&](std::size_t iStep) {
  const float tau = 0.55;
  
  lattice.apply<BgkCollideO>(bulk_list, tau);
  lattice.apply<BounceBackO>(wall_list);
  lattice.apply<BouzidiO>(bouzidi.getCount(), bouzidi.getConfig());
  
  lattice.stream();
});
}
